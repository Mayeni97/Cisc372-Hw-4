#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "timer.h"
#include "fractal.h"

__global__ void computeFractalKernel(unsigned char *pic, int width, int height, int num_frames, double xMid, double yMid, double delta) {
    int frame = blockIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
        double aspect_ratio = (double)width / height;
        double x0 = xMid - delta * aspect_ratio;
        double y0 = yMid - delta;
        double dx = 2.0 * delta * aspect_ratio / width;
        double dy = 2.0 * delta / height;

        double cx = x0 + col * dx;
        double cy = y0 + row * dy;

        double x = cx;
        double y = cy;
        int depth = 256;

        double x2, y2;
        do {
            x2 = x * x;
            y2 = y * y;
            y = 2 * x * y + cy;
            x = x2 - y2 + cx;
            depth--;
        } while ((depth > 0) && ((x2 + y2) < 5.0));

        pic[frame * height * width + row * width + col] = (unsigned char)depth;
    }
}

static const double Delta = 0.001;
static const double xMid =  0.23701;
static const double yMid =  0.521;

int main(int argc, char *argv[]) {
  double start, end;

  printf("Fractal v1.6 [serial]\n");

  /* read command line arguments */
  if (argc != 4) {fprintf(stderr, "usage: %s height width num_frames\n", argv[0]); exit(-1);}
  int width = atoi(argv[1]);
  if (width < 10) {fprintf(stderr, "error: width must be at least 10\n"); exit(-1);}
  int height = atoi(argv[2]);
  if (height < 10) {fprintf(stderr, "error: height must be at least 10\n"); exit(-1);}
  int num_frames = atoi(argv[3]);
  if (num_frames < 1) {fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);}
  printf("Computing %d frames of %d by %d fractal\n", num_frames, width, height);

  /* allocate image array */
  unsigned char *h_pic = (unsigned char *)malloc(num_frames * height * width * sizeof(unsigned char));

  // Allocate memory
  unsigned char *d_pic;
  hipMalloc((void **)&d_pic, num_frames * height * width * sizeof(unsigned char));

  /* start time */
  GET_TIME(start);

  // Define block and grid dimensions
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (height + threadsPerBlock.y - 1) / threadsPerBlock.y,
                 num_frames);

  // Launch CUDA kernel
  computeFractalKernel<<<numBlocks, threadsPerBlock>>>(d_pic, width, height, num_frames, xMid, yMid, Delta);

  // Copy results back to host
  hipMemcpy(h_pic, d_pic, num_frames * height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);

  /* end time */
  GET_TIME(end);
  double elapsed = end - start;
  printf("CUDA compute time: %.4f s\n", elapsed);

  /* write frames to BMP files */
  if ((width <= 320) && (num_frames <= 100)) { /* do not write if images large or many */
    for (int frame = 0; frame < num_frames; frame++) {
      char name[32];
      sprintf(name, "fractal%d.bmp", frame + 1000);
      writeBMP(width, height, &h_pic[frame * height * width], name);
    }
  }

  // Free memory
  free(h_pic);
  hipFree(d_pic);

  return 0;
} /* main */
